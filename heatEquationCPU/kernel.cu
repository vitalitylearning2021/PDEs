﻿#include <stdio.h>

#include <stdlib.h>

#include "InputOutput.cuh"

#define Npixels	640							// --- Image width
#define Mpixels	640							// --- Image height

#define MAX_NUM_ITERS	50000			// --- Maximum number of FDTD iterations. Must be even.

// --- struct ProblemParameters containing all the problem parameters
typedef struct {
	int		x, y;						// --- Coordinates of the pipe center
	float	radius;						// --- Radius of the pipe
	float	T_pipe, T_air, T_ground;	// --- Temperatures in pipe, air, ground
} ProblemParameters;

ProblemParameters bc = { Npixels / 2, Mpixels / 2, Npixels / 10.f, 200.f, 20.f, 10.f };

/***************************************************/
/* SETTING THE BOUNDARY CONDITIONS DEVICE FUNCTION */
/***************************************************/
void setBoundaryConditions(float* d_T, const int width, const int height, const int idx,
	const int tidx, const int tidy, const ProblemParameters bc) {

	// --- Set the pipe temperature to T_pipe and return
	float distanceFromPipeCenterSquared = ((tidx - bc.x) * (tidx - bc.x) + (tidy - bc.y) * (tidy - bc.y));
	if (distanceFromPipeCenterSquared < bc.radius * bc.radius) d_T[idx] = bc.T_pipe;

	// --- Set the left, right and upper border temperature to T_air and return
	if ((tidx == 0) || (tidx == width - 1) || (tidy == 0)) d_T[idx] = bc.T_air;

	// --- Set the lower border temperature to T_ground and return
	if (tidy == height - 1) d_T[idx] = bc.T_ground;

}

/*************************/
/* RESET TEMPERATURE CPU */
/*************************/
void resetTemperatureCPU(float* h_temperature, int width, int height, ProblemParameters bc) {

	for (int j = 0; j < height; j++)
		for (int i = 0; i < width; i++) {
			const int idx = j * width + i;
			h_temperature[idx] = bc.T_air;
		}
}

/***********************************/
/* JACOBI ITERATION FUNCTION - CPU */
/***********************************/
void temperatureUpdateCPU(float* __restrict h_T, float* __restrict h_T_new, const int width, const int height, const ProblemParameters bc) {

	// --- Only update "interior" (not boundary) node points
	for (int j = 0; j < height; j++)
		for (int i = 0; i < width; i++) {
			const int idx = j * width + i;
			if ((i > 0) && (i < width - 1) && (j > 0) && (j < height - 1))
				h_T_new[idx] = 0.25f * (h_T[idx - 1] +
					h_T[idx + 1] +
					h_T[idx + width] +
					h_T[idx - width]);
			setBoundaryConditions(h_T_new, width, height, idx, i, j, bc);
		}
}

/********/
/* MAIN */
/********/
int main() {

	float* h_temperature = (float*)malloc(Npixels * Mpixels * sizeof(float));
	float* h_temperature_new = (float*)malloc(Npixels * Mpixels * sizeof(float));

	resetTemperatureCPU(h_temperature, Npixels, Mpixels, bc);

	for (int iter = 0; iter < MAX_NUM_ITERS / 2; iter++) {
		temperatureUpdateCPU(h_temperature, h_temperature_new, Npixels, Mpixels, bc);
		temperatureUpdateCPU(h_temperature_new, h_temperature, Npixels, Mpixels, bc);
	}

	saveCPUrealtxt(h_temperature, ".\\CPU_result.txt", Npixels * Mpixels);

	return 0;
}